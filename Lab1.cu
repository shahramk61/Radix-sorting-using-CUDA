#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdlib.h>
#include <stdio.h>
#include <iostream>
#include <chrono>


#define size_data 32
#define upper_bit 6
#define lower_bit 0




__global__ void Radix(int* keys, int* values) {



	__shared__ volatile int keys_1[size_data * 2];
	__shared__ volatile int values_1[size_data * 2];



	keys_1[threadIdx.x] = keys[threadIdx.x];
	values_1[threadIdx.x] = values[threadIdx.x];

	unsigned int bit_mask = 1 << lower_bit;
	unsigned int offset = 0;
	unsigned int mask = 0xFFFFFFFFU << threadIdx.x;
	unsigned int pos_index;

	for (int i = lower_bit; i <= upper_bit; i++) {



		int temp_keys = keys_1[((size_data - 1) - threadIdx.x) + offset];
		int temp_values = values_1[((size_data - 1) - threadIdx.x) + offset];
		unsigned int current_bit = temp_keys&bit_mask;


		unsigned int ones = __ballot(current_bit);
		unsigned int zeroes = ~ones;


		offset ^= size_data;

		if (!current_bit)
		{
			pos_index = __popc(zeroes&mask);
		}
		else {
			pos_index = __popc(zeroes) + __popc(ones&mask);
		}

		keys_1[pos_index - 1 + offset] = temp_keys;
		values_1[pos_index - 1 + offset] = temp_values;

		bit_mask <<= 1;
	}


	// copy back the result
	keys[threadIdx.x] = keys_1[threadIdx.x + offset];
	// copy back the result
	values[threadIdx.x] = values_1[threadIdx.x + offset];


}


__global__ void rle(int* d_data, int* d_run, int* d_Cell_id) {

	int i = threadIdx.x;
	unsigned int laneid = i & 0x1f;
	int val = d_data[i];
	int nval = __shfl_down(val, 1);
	unsigned int mask = __ballot(nval != val);
	int offset = __popc(mask & ((1 << laneid) - 1));
	int zcnt = __clz(mask & ((1 << laneid) - 1));
	int runcnt = zcnt - 31 + laneid;


	if (nval != val || i == 31) {

		d_Cell_id[offset] = val;
		d_run[offset] = runcnt;

	}


}

int main() {



	
	int NUMBER_OF_PARTICLES = 32;
	int NUMBER_OF_KEYS = 16;
	int keys[32];
	int value[32];
	int *d_keys;
	int *d_values;
	int *d_Count;
	int *d_Cell_id;
	int Count[16];
	int Cell_id[16];
	int offset = 0;
	int counter = 0;

	srand(time(NULL));
	for (int i = 0; i < NUMBER_OF_PARTICLES; i++) {
		keys[i] = rand() % NUMBER_OF_KEYS;
		value[i] = i;
	}


	//allocate memory for keys
	hipMalloc((void**)&d_keys, NUMBER_OF_PARTICLES * sizeof(int));
	//copy keys to shares memory
	hipMemcpy(d_keys, keys, NUMBER_OF_PARTICLES * sizeof(int), hipMemcpyHostToDevice);
	//allocate memory for values
	hipMalloc((void**)&d_values, NUMBER_OF_PARTICLES * sizeof(int));
	//copy values to shared memory
	hipMemcpy(d_values, value, NUMBER_OF_PARTICLES * sizeof(int), hipMemcpyHostToDevice);

	Radix << <1, NUMBER_OF_PARTICLES >> > (d_keys, d_values);
	hipMalloc((void**)&d_Count, NUMBER_OF_KEYS * sizeof(int));
	hipMalloc((void**)&d_Cell_id, NUMBER_OF_KEYS * sizeof(int));

	rle << <1, NUMBER_OF_PARTICLES >> > (d_keys, d_Count, d_Cell_id);

	hipMemcpy(value, d_values, NUMBER_OF_PARTICLES * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(Cell_id, d_Cell_id, NUMBER_OF_KEYS * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(Count, d_Count, NUMBER_OF_KEYS * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(keys, d_keys, NUMBER_OF_PARTICLES * sizeof(int), hipMemcpyDeviceToHost);
	hipFree(d_keys);
	hipFree(d_values);
	hipFree(d_Count);

	for (int i = 0; i < NUMBER_OF_KEYS; i++) {
		if (i != Cell_id[counter]) {// empty cell

			printf("Cell id: %2d  Count: %d\n", i, 0);
		}
		else {

			printf("Cell id: %2d  Count: %d	Particle id: ", Cell_id[counter], Count[counter]);

			for (int k = 0; k < Count[counter]; k++) {

				printf(" %d", value[offset]);
				offset += 1;
			}
			printf("\n");
			counter += 1;

		}








	}

	return 0;
}
